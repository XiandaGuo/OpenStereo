#include "hip/hip_runtime.h"
//#include <torch/torch.h>
#include <torch/extension.h>
//#include <torch/serialize/tensor.h>
//#include <ATen/ATen.h>
//#include <ATen/cuda/HIPContext.h>

#define CUDA_NUM_THREADS 256 
#define THREADS_PER_BLOCK 64 

#define DIM0(TENSOR) ((TENSOR).x)
#define DIM1(TENSOR) ((TENSOR).y)
#define DIM2(TENSOR) ((TENSOR).z)
#define DIM3(TENSOR) ((TENSOR).w)

#define DIM3_INDEX(TENSOR, xx, yy, zz, ww) ((TENSOR)[((xx) * (TENSOR##_stride.x)) + ((yy) * (TENSOR##_stride.y)) + ((zz) * (TENSOR##_stride.z)) + ((ww) * (TENSOR##_stride.w))])

#ifdef __cplusplus
    extern "C" {
#endif



__global__ void Max (const int n, const float *top_temp, float *top_data, float *mask,
     const int mask_index){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  if (top_data[index] < top_temp[index])
    {
      top_data[index] = top_temp[index];
      mask[index] = mask_index;
    }
}

__global__ void get_temp_grad (const int n, const float *gradOutput, const float *mask,
	       float *top_grad, const int mask_index){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  if (((int) mask[index]) == mask_index)
    top_grad[index] = gradOutput[index];
}

__global__ void MaxDepth (const int n, const float *bottom_data, const int step,
	  const int depth, float *idx){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int base = index / step * step * depth + index % step;
  int k = 0;
  for (int i = 1; i < depth; i++)
    if (bottom_data[base + k * step] < bottom_data[base + i * step])
      k = i;
  idx[index] = k;
}

__global__ void sga_down_forward (const int n, const float *filters, const int height,
		  const int width, const int depth, const int wsize,
		  float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
//  int wsize=radius+1;
  int base = index / width * step * depth + index % width;	//up->down
  int fbase = index / width * step * wsize + index % width;

  int kp = 0;

  for (int row = 0; row < height; row++)
    {
      int shift = fbase + row * width;

      int base0 = base + row * width;
      int k = kp;
      kp = 0;

/*        if(row-1>=0)
            for(int i = 1; i < depth; i++){
	        if(top_data[base0-width+k*step]<top_data[base0-width+i*step])
		    k = i;
*/
      for (int d = 0; d < depth; d++)
	{
	  float temp = 0;
	  int location = base0 + d * step;
	  temp += top_data[location] * filters[shift];
	  if (row - 1 >= 0)
	    temp += top_data[location - width] * filters[shift + step];
	  else
	    temp += top_data[location] * filters[shift + step];

	  if (row - 1 >= 0 && d - 1 >= 0)
	    temp +=
	      top_data[location - width - step] * filters[shift + 2 * step];
	  else
	    temp += top_data[location] * filters[shift + 2 * step];
	  if (row - 1 >= 0 && d + 1 < depth)
	    temp +=
	      top_data[location - width + step] * filters[shift + 3 * step];
	  else
	    temp += top_data[location] * filters[shift + 3 * step];
	  if (row - 1 >= 0)
	    temp +=
	      top_data[base0 - width + k * step] * filters[shift + 4 * step];
	  else
	    temp += top_data[location] * filters[shift + 4 * step];
	  top_data[location] = temp;

	  if (top_data[base0 + kp * step] < temp)
	    kp = d;

	}
    }
}

__global__ void sga_down_data_backward (const int n, const float *filters, float *top_diff,
			const float *idx, const int height, const int width,
			const int depth, const int wsize, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / width * step * depth + index % width;	//up->down
  int fbase = index / width * step * wsize + index % width;
//1
  int base_idx = index / width * step + index % width;
//
  for (int row = height - 1; row >= 0; row--)
    {
      int shift = fbase + row * width;
      for (int d = 0; d < depth; d++)
	{
	  int location = base + d * step + row * width;
	  float temp = top_diff[location];
	  if (row + 1 < height)
	    temp +=
	      top_diff[location + width] * filters[shift + width + step];

	  if (row + 1 < height && d + 1 < depth)
	    temp +=
	      top_diff[location + width + step] * filters[shift + width +
							  2 * step];
	  if (row + 1 < height && d - 1 >= 0)
	    temp +=
	      top_diff[location + width - step] * filters[shift + width +
							  3 * step];
	  top_diff[location] = temp;
	  bottom_diff[location] += temp * filters[shift];
	}
//2
      if (row + 1 < height)
	{
	  int k = idx[base_idx + row * width];
	  int location = base + k * step + row * width;
	  float temp = 0;
	  for (int d = 0; d < depth; d++)
	    temp +=
	      top_diff[base + row * width + width +
		       d * step] * filters[shift + width + 4 * step];
	  top_diff[location] += temp;
	  bottom_diff[location] += temp * filters[shift];
	}
//2

    }

/*	for(int d = 0; d < depth; d ++){
		int shift = fbase;
		int location = base + d * step;
		bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]); 
 //       bottom_diff[location] += top_diff[location];
		shift += width;
		location += width;
		bottom_diff[location] += top_diff[location] * filters[shift + 2*step];	
	}
	for(int row=1;row<height;row++){
		int location = base + row * width;
		int shift = fbase + row * width;
		bottom_diff[location] += top_diff[location] * filters[shift + 3*step]; 
		location += (depth - 1)*step;
		bottom_diff[location] += top_diff[location] * filters[shift + 4*step]; 
	}
*/
  for (int row = 0; row < height; row++)
    {
      int location = base + row * width;
      int shift = fbase + row * width;
      bottom_diff[location] += top_diff[location] * filters[shift + 2 * step];
      location += (depth - 1) * step;
      bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
    }
}

__global__ void sga_down_weight_backward (const int n, const float *bottom_data,
			  const float *top_data, const float *temp_diff,
			  const float *idx, const int height, const int width,
			  const int depth, const int wsize,
			  float *filters_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / step * step * depth + index % step;	//up->down
  int fbase = index / step * step * wsize + index % step;

  int row = index % step / width;
  for (int i = 0; i < depth; i++)
    filters_diff[fbase] +=
      temp_diff[base + i * step] * bottom_data[base + i * step];
  if (row - 1 >= 0)
    {
      int location = fbase + step;
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + i * step - width];

      location = fbase + 2 * step;
      filters_diff[location] += temp_diff[base] * bottom_data[base];
      for (int i = 1; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i - 1) * step -
						width];

      location = fbase + 3 * step;
      filters_diff[location] +=
	temp_diff[base + (depth - 1) * step] * bottom_data[base +
							   (depth -
							    1) * step];
      for (int i = 0; i < depth - 1; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i + 1) * step -
						width];
    }
/*
    else{
		for(int i=0; i<depth; i++){
			float temp = temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		    filters_diff[fbase + 3*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + 4*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		}

	}
*/
//1
  if (row - 1 >= 0)
    {
      int location = fbase + 4 * step;
      int k = idx[index - width];
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + k * step - width];
    }
//
/*
    else{
		int location = fbase + 2*step;
		for(int i=0; i<depth; i++)
			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}
*/
}



__global__ void sga_up_forward (const int n, const float *filters, const int height,
		const int width, const int depth, const int wsize,
		float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= n)
    {
      return;
    }
  int step = height * width;
  //   int wsize=radius+1;

  int base = index / width * step * depth + index % width;	//up->down
  int fbase = index / width * step * wsize + index % width;

  int kp = 0;			//1

  for (int row = height - 1; row >= 0; row--)
    {
      int shift = fbase + row * width;
//2
      int base0 = base + row * width;
      int k = kp;
      kp = 0;
//2
      for (int d = 0; d < depth; d++)
	{
	  float temp = 0;
	  int location = base + d * step + row * width;
	  temp += top_data[location] * filters[shift];
	  if (row + 1 < height)
	    temp += top_data[location + width] * filters[shift + step];
	  else
	    temp += top_data[location] * filters[shift + step];

	  if (row + 1 < height && d - 1 >= 0)
	    temp +=
	      top_data[location + width - step] * filters[shift + 2 * step];
	  else
	    temp += top_data[location] * filters[shift + 2 * step];
	  if (row + 1 < height && d + 1 < depth)
	    temp +=
	      top_data[location + width + step] * filters[shift + 3 * step];
	  else
	    temp += top_data[location] * filters[shift + 3 * step];

//3
	  if (row + 1 < height)
	    temp +=
	      top_data[base0 + width + k * step] * filters[shift + 4 * step];
	  else
	    temp += top_data[location] * filters[shift + 4 * step];
	  top_data[location] = temp;

	  if (top_data[base0 + kp * step] < temp)
	    kp = d;
//3

	}
    }
}

__global__ void sga_up_data_backward (const int n, const float *filters, float *top_diff,
		      const float *idx, const int height, const int width,
		      const int depth, const int wsize, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / width * step * depth + index % width;	//up->down
  int fbase = index / width * step * wsize + index % width;

//1
  int base_idx = index / width * step + index % width;
//
  for (int row = 0; row < height; row++)
    {
      int shift = fbase + row * width;
      for (int d = 0; d < depth; d++)
	{
	  int location = base + d * step + row * width;
	  float temp = top_diff[location];
	  if (row - 1 >= 0)
	    temp +=
	      top_diff[location - width] * filters[shift - width + step];
	  if (row - 1 >= 0 && d + 1 < depth)
	    temp +=
	      top_diff[location - width + step] * filters[shift - width +
							  2 * step];
	  if (row - 1 >= 0 && d - 1 >= 0)
	    temp +=
	      top_diff[location - width - step] * filters[shift - width +
							  3 * step];
	  top_diff[location] = temp;
	  bottom_diff[location] += temp * filters[shift];
	}

//2
      if (row - 1 >= 0)
	{
	  int k = idx[base_idx + row * width];
	  int location = base + k * step + row * width;
	  float temp = 0;
	  for (int d = 0; d < depth; d++)
	    temp +=
	      top_diff[base + row * width - width +
		       d * step] * filters[shift - width + 4 * step];
	  top_diff[location] += temp;
	  bottom_diff[location] += temp * filters[shift];
	}
//2
    }

/*	for(int d = 0; d < depth; d ++){
		int shift = fbase + width*(height-1);
		int location = base + width*(height-1) + d * step;
		bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]); 
//        bottom_diff[location] += top_diff[location];
		shift -= width;
		location -= width;
		bottom_diff[location] += top_diff[location] * filters[shift + 2*step];	
	}
	for(int row=0;row<height-1;row++){
		int shift = fbase + row * width;
		int location = base + row * width;
		bottom_diff[location] += top_diff[location] * filters[shift + 3*step]; 
		location += (depth - 1)*step;
		bottom_diff[location] += top_diff[location] * filters[shift + 4*step]; 
	}*/
  for (int row = 0; row < height; row++)
    {
      int shift = fbase + row * width;
      int location = base + row * width;
      bottom_diff[location] += top_diff[location] * filters[shift + 2 * step];
      location += (depth - 1) * step;
      bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
    }
}

__global__ void sga_up_weight_backward (const int n, const float *bottom_data,
			const float *top_data, const float *temp_diff,
			const float *idx, const int height, const int width,
			const int depth, const int wsize, float *filters_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / step * step * depth + index % step;	//up->down
  int fbase = index / step * step * wsize + index % step;

  int row = index % step / width;
  for (int i = 0; i < depth; i++)
    filters_diff[fbase] +=
      temp_diff[base + i * step] * bottom_data[base + i * step];
  if (row + 1 < height)
    {
      int location = fbase + step;
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + i * step + width];

      location = fbase + 2 * step;
      filters_diff[location] += temp_diff[base] * bottom_data[base];
      for (int i = 1; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i - 1) * step +
						width];

      location = fbase + 3 * step;
      filters_diff[location] +=
	temp_diff[base + (depth - 1) * step] * bottom_data[base +
							   (depth -
							    1) * step];
      for (int i = 0; i < depth - 1; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i + 1) * step +
						width];
    }
/*
    else{
		//int location = fbase + step;
		for(int i=0; i<depth; i++){
			float temp = temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		    filters_diff[fbase + 3*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + 4*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		}
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];		
//		location = fbase + 3*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
//		
//		location = fbase + 4*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
//1
  if (row + 1 < height)
    {
      int location = fbase + 4 * step;
      int k = idx[index + width];
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + k * step + width];
    }
//

/*
    else{
		int location = fbase + 2*step;
		for(int i=0; i<depth; i++)
			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
}

__global__ void sga_right_forward (const int n, const float *filters, const int height,
		   const int width, const int depth, const int wsize,
		   float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= n)
    {
      return;
    }
  int step = height * width;
  //   int wsize=radius+1;

  int base = index / height * step * depth + (index % height) * width;	//up->down
  int fbase = index / height * step * wsize + (index % height) * width;

  int kp = 0;

  for (int col = 0; col < width; col++)
    {
      int shift = fbase + col;
//2
      int base0 = base + col;
      int k = kp;
      kp = 0;
//2
      for (int d = 0; d < depth; d++)
	{
	  float temp = 0;
	  int location = base + d * step + col;
	  temp += top_data[location] * filters[shift];
	  if (col - 1 >= 0)
	    temp += top_data[location - 1] * filters[shift + step];
	  else
	    temp += top_data[location] * filters[shift + step];

	  if (col - 1 >= 0 && d - 1 >= 0)
	    temp += top_data[location - 1 - step] * filters[shift + 2 * step];
	  else
	    temp += top_data[location] * filters[shift + 2 * step];
	  if (col - 1 >= 0 && d + 1 < depth)
	    temp += top_data[location - 1 + step] * filters[shift + 3 * step];
	  else
	    temp += top_data[location] * filters[shift + 3 * step];

//3
	  if (col - 1 >= 0)
	    temp +=
	      top_data[base0 - 1 + k * step] * filters[shift + 4 * step];
	  else
	    temp += top_data[location] * filters[shift + 4 * step];
	  top_data[location] = temp;

	  if (top_data[base0 + kp * step] < temp)
	    kp = d;
//3
	}
    }
}

__global__ void sga_right_data_backward (const int n, const float *filters, float *top_diff,
			 const float *idx, const int height, const int width,
			 const int depth, const int wsize, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / height * step * depth + (index % height) * width;	//up->down
  int fbase = index / height * step * wsize + (index % height) * width;
//1
  int base_idx = index / height * step + (index % height) * width;
//
  for (int col = width - 1; col >= 0; col--)
    {
      int shift = fbase + col;
      for (int d = 0; d < depth; d++)
	{
	  int location = base + d * step + col;
	  float temp = top_diff[location];
	  if (col + 1 < width)
	    temp += top_diff[location + 1] * filters[shift + 1 + step];
	  if (col + 1 < width && d + 1 < depth)
	    temp +=
	      top_diff[location + 1 + step] * filters[shift + 1 + 2 * step];
	  if (col + 1 < width && d - 1 >= 0)
	    temp +=
	      top_diff[location + 1 - step] * filters[shift + 1 + 3 * step];
	  top_diff[location] = temp;
	  bottom_diff[location] += (temp * filters[shift]);
	}
//2
      if (col + 1 < width)
	{
	  int k = idx[base_idx + col];
	  int location = base + k * step + col;
	  float temp = 0;
	  for (int d = 0; d < depth; d++)
	    temp +=
	      top_diff[base + col + 1 + d * step] * filters[shift + 1 +
							    4 * step];
	  top_diff[location] += temp;
	  bottom_diff[location] += temp * filters[shift];
	}
//2     
    }
/*
	for(int d = 0; d < depth; d ++){
		int shift = fbase;// + width*(height-1);
		int location = base;// + width*(height-1) + d * step;
		bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]);
 //       bottom_diff[location] += top_diff[location];
		shift += 1;
		location += 1;
		bottom_diff[location] += top_diff[location] * filters[shift + 2*step];	
	}
	for(int col=1;col<width;col++){
		int shift = fbase + col;
		int location = base + col;
		bottom_diff[location] += top_diff[location] * filters[shift + 3*step]; 
		location += (depth - 1)*step;
		bottom_diff[location] += top_diff[location] * filters[shift + 4*step]; 
	}*/
  for (int col = 0; col < width; col++)
    {
      int shift = fbase + col;
      int location = base + col;
      bottom_diff[location] += top_diff[location] * filters[shift + 2 * step];
      location += (depth - 1) * step;
      bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
    }
}

__global__ void sga_right_weight_backward (const int n, const float *bottom_data,
			   const float *top_data, const float *temp_diff,
			   const float *idx, const int height,
			   const int width, const int depth, const int wsize,
			   float *filters_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / step * step * depth + index % step;	//up->down
  int fbase = index / step * step * wsize + index % step;

  //   int row = index%step/width;
  int col = index % step % width;
  for (int i = 0; i < depth; i++)
    filters_diff[fbase] +=
      temp_diff[base + i * step] * bottom_data[base + i * step];
  if (col - 1 >= 0)
    {
      int location = fbase + step;
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + i * step - 1];

      location = fbase + 2 * step;
      filters_diff[location] += temp_diff[base] * bottom_data[base];
      for (int i = 1; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i - 1) * step - 1];

      location = fbase + 3 * step;
      filters_diff[location] +=
	temp_diff[base + (depth - 1) * step] * bottom_data[base +
							   (depth -
							    1) * step];
      for (int i = 0; i < depth - 1; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i + 1) * step - 1];
    }
/*
    else{
		//int location = fbase + step;
		for(int i=0; i<depth; i++){
			float temp = temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		    filters_diff[fbase + 3*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + 4*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		}
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];		
//		location = fbase + 3*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
//		
//		location = fbase + 4*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
//1
  if (col - 1 >= 0)
    {
      int location = fbase + 4 * step;
      int k = idx[index - 1];
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + k * step - 1];
    }
//
/*
    else{
		int location = fbase + 2*step;
		for(int i=0; i<depth; i++)
			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
}

__global__ void sga_left_forward (const int n, const float *filters, const int height,
		  const int width, const int depth, const int wsize,
		  float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= n)
    {
      return;
    }
  int step = height * width;
  //   int wsize=radius+1;

  int base = index / height * step * depth + (index % height) * width;	//up->down
  int fbase = index / height * step * wsize + (index % height) * width;

  int kp = 0;

  for (int col = width - 1; col >= 0; col--)
    {
      int shift = fbase + col;
//2
      int base0 = base + col;
      int k = kp;
      kp = 0;
//2
      for (int d = 0; d < depth; d++)
	{
	  float temp = 0;
	  int location = base + d * step + col;
	  temp += top_data[location] * filters[shift];
	  if (col + 1 < width)
	    temp += top_data[location + 1] * filters[shift + step];
	  else
	    temp += top_data[location] * filters[shift + step];

	  if (col + 1 < width && d - 1 >= 0)
	    temp += top_data[location + 1 - step] * filters[shift + 2 * step];
	  else
	    temp += top_data[location] * filters[shift + 2 * step];
	  if (col + 1 < width && d + 1 < depth)
	    temp += top_data[location + 1 + step] * filters[shift + 3 * step];
	  else
	    temp += top_data[location] * filters[shift + 3 * step];

//3
	  if (col + 1 < width)
	    temp +=
	      top_data[base0 + 1 + k * step] * filters[shift + 4 * step];
	  else
	    temp += top_data[location] * filters[shift + 4 * step];
	  top_data[location] = temp;

	  if (top_data[base0 + kp * step] < temp)
	    kp = d;
//3
	}
    }
}

__global__ void sga_left_data_backward (const int n, const float *filters, float *top_diff,
			const float *idx, const int height, const int width,
			const int depth, const int wsize, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / height * step * depth + (index % height) * width;	//up->down
  int fbase = index / height * step * wsize + (index % height) * width;
//1
  int base_idx = index / height * step + (index % height) * width;
//
  for (int col = 0; col < width; col++)
    {
      int shift = fbase + col;
      for (int d = 0; d < depth; d++)
	{
	  int location = base + d * step + col;
	  float temp = top_diff[location];
	  if (col - 1 >= 0)
	    temp += top_diff[location - 1] * filters[shift - 1 + step];
	  if (col - 1 >= 0 && d + 1 < depth)
	    temp +=
	      top_diff[location - 1 + step] * filters[shift - 1 + 2 * step];
	  if (col - 1 >= 0 && d - 1 >= 0)
	    temp +=
	      top_diff[location - 1 - step] * filters[shift - 1 + 3 * step];
	  top_diff[location] = temp;
	  bottom_diff[location] += temp * filters[shift];
	}
//2
      if (col - 1 >= 0)
	{
	  int k = idx[base_idx + col];
	  int location = base + k * step + col;
	  float temp = 0;
	  for (int d = 0; d < depth; d++)
	    temp +=
	      top_diff[base + col - 1 + d * step] * filters[shift - 1 +
							    4 * step];
	  top_diff[location] += temp;
//top_diff[base + col - 1 + d*step] * filters[shift - 1 + 4*step];
	  bottom_diff[location] += temp * filters[shift];
	}
//2             
    }
/*
	for(int d = 0; d < depth; d ++){
		int shift = fbase + width-1;// + width*(height-1);
		int location = base + width-1;// + width*(height-1) + d * step;
		bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]); 
//        bottom_diff[location] += top_diff[location];
		shift -= 1;
		location -= 1;
		bottom_diff[location] += top_diff[location] * filters[shift + 2*step];	
	}
	for(int col=0;col<width-1;col++){
		int shift = fbase + col;
		int location = base + col;
		bottom_diff[location] += top_diff[location] * filters[shift + 3*step]; 
		location += (depth - 1)*step;
		bottom_diff[location] += top_diff[location] * filters[shift + 4*step]; 
	}*/
  for (int col = 0; col < width; col++)
    {
      int shift = fbase + col;
      int location = base + col;
      bottom_diff[location] += top_diff[location] * filters[shift + 2 * step];
      location += (depth - 1) * step;
      bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
    }
}

__global__ void sga_left_weight_backward (const int n, const float *bottom_data,
			  const float *top_data, const float *temp_diff,
			  const float *idx, const int height, const int width,
			  const int depth, const int wsize,
			  float *filters_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / step * step * depth + index % step;	//up->down
  int fbase = index / step * step * wsize + index % step;

  //   int row = index%step/width;
  int col = index % step % width;
  for (int i = 0; i < depth; i++)
    filters_diff[fbase] +=
      temp_diff[base + i * step] * bottom_data[base + i * step];
  if (col + 1 < width)
    {
      int location = fbase + step;
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + i * step + 1];

      location = fbase + 2 * step;
      filters_diff[location] += temp_diff[base] * bottom_data[base];
      for (int i = 1; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i - 1) * step + 1];

      location = fbase + 3 * step;
      filters_diff[location] +=
	temp_diff[base + (depth - 1) * step] * bottom_data[base +
							   (depth -
							    1) * step];
      for (int i = 0; i < depth - 1; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i + 1) * step + 1];
    }
/*
    else{
		//int location = fbase + step;
		for(int i=0; i<depth; i++){
			float temp = temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		    filters_diff[fbase + 3*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + 4*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		}
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];		
//		location = fbase + 3*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
//		
//		location = fbase + 4*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
//1
  if (col + 1 < width)
    {
      int location = fbase + 4 * step;
      int k = idx[index + 1];
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + k * step + 1];
    }
//
/*
    else{
		int location = fbase + 2*step;
		for(int i=0; i<depth; i++)
			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}
*/
}

void sga_kernel_forward (at::Tensor input, at::Tensor guidance_down,
		    at::Tensor guidance_up, at::Tensor guidance_right,
		    at::Tensor guidance_left, at::Tensor temp_out,
		    at::Tensor output, at::Tensor mask){

  int num = input.size(0);
  int channel = input.size(1);
  int depth = input.size(2);
  int height = input.size(3);
  int width = input.size(4);
  int wsize = guidance_down.size(2);

  //THCudaTensor_nElement(state, input);
  float *top_data = output.data<float>();
  float *top_temp = temp_out.data<float>();
  float *top_mask = mask.data<float>();

  const float *bottom_data = input.data<float>();
  const float *g0 = guidance_down.data<float>();
  const float *g1 = guidance_up.data<float>();
  const float *g2 = guidance_right.data<float>();
  const float *g3 = guidance_left.data<float>();

  int n = num * channel * width;
  int threads = (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
  int N = input.numel ();
//      hipStream_t stream = at::cuda::getCurrentCUDAStream(); 
  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_down_forward <<< threads, CUDA_NUM_THREADS >>> (n, g0, height, width,
						      depth, wsize, top_temp);
//      hipMemset( top_mask, 0, sizeof(float)*N);
  hipMemcpy (top_data, top_temp, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);

  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_up_forward <<< threads, CUDA_NUM_THREADS >>> (n, g1, height, width,
						    depth, wsize, top_temp);

  Max <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, top_data, top_mask, 1);

  n = num * channel * height;
  threads = (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;

  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_right_forward <<< threads, CUDA_NUM_THREADS >>> (n, g2, height, width,
						       depth, wsize,
						       top_temp);
  Max <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, top_data, top_mask, 2);

  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_left_forward <<< threads, CUDA_NUM_THREADS >>> (n, g3, height, width,
						      depth, wsize, top_temp);
  Max <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, top_data, top_mask, 3);

//      hipMemset( top_temp, 0, sizeof(float)*THCudaTensor_nElement(state, top_temp));

}

void sga_kernel_backward (at::Tensor input, at::Tensor guidance_down,
		     at::Tensor guidance_up, at::Tensor guidance_right,
		     at::Tensor guidance_left, at::Tensor temp_out,
		     at::Tensor mask, at::Tensor max_idx,
		     at::Tensor gradOutput, at::Tensor temp_grad,
		     at::Tensor gradInput, at::Tensor grad_down,
		     at::Tensor grad_up, at::Tensor grad_right,
		     at::Tensor grad_left){

  int num = input.size(0);
  int channel = input.size(1);
  int depth = input.size(2);
  int height = input.size(3);
  int width = input.size(4);
  int wsize = guidance_down.size(2);

  //THCudaTensor_nElement(state, input);
  float *top_grad = temp_grad.data<float>();
  float *top_temp = temp_out.data<float>();
  const float *top_mask = mask.data<float>();

  const float *bottom_data = input.data<float>();
  const float *grad_out = gradOutput.data<float>();

  const float *g0 = guidance_down.data<float>();
  const float *g1 = guidance_up.data<float>();
  const float *g2 = guidance_right.data<float>();
  const float *g3 = guidance_left.data<float>();

  float *grad0 = grad_down.data<float>();
  float *grad1 = grad_up.data<float>();
  float *grad2 = grad_right.data<float>();
  float *grad3 = grad_left.data<float>();
  float *grad_input = gradInput.data<float>();

  float *idx = max_idx.data<float>();

  int N = input.numel ();
//      hipStream_t stream = at::cuda::getCurrentCUDAStream(); 

//backward for left             
  int n = num * channel * height;
//              hipMemcpy(top_temp, bottom_data, sizeof(float)*N, hipMemcpyDeviceToDevice);
//              sga_left_forward<<<(n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>
//              (n,g3,height,width,depth,wsize,top_temp);

  hipMemset (top_grad, 0, sizeof (float) * N);
  get_temp_grad <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, grad_out, top_mask, top_grad, 3);

  N = num * channel * width * height;
  MaxDepth <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, height * width, depth, idx);

  sga_left_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g3, top_grad, idx, height, width, depth, wsize,
			  grad_input);
  n = num * channel * width * height;
  sga_left_weight_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, bottom_data, top_temp, top_grad, idx, height,
			  width, depth, wsize, grad3);
//backward for down             
  N = input.numel ();
  n = num * channel * width;
  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_down_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g0, height, width, depth, wsize, top_temp);

  hipMemset (top_grad, 0, sizeof (float) * N);
  get_temp_grad <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, grad_out, top_mask, top_grad, 0);

  N = num * channel * width * height;
  MaxDepth <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, height * width, depth, idx);

  sga_down_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g0, top_grad, idx, height, width, depth, wsize,
			  grad_input);
  n = num * channel * width * height;
  sga_down_weight_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, bottom_data, top_temp, top_grad, idx, height,
			  width, depth, wsize, grad0);
// backward for up              
  N = input.numel ();
  n = num * channel * width;
  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_up_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g1, height, width, depth, wsize, top_temp);

  hipMemset (top_grad, 0, sizeof (float) * N);
  get_temp_grad <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, grad_out, top_mask, top_grad, 1);
  N = num * channel * width * height;
  MaxDepth <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, height * width, depth, idx);

  sga_up_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g1, top_grad, idx, height, width, depth, wsize,
			  grad_input);
  n = num * channel * width * height;
  sga_up_weight_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, bottom_data, top_temp, top_grad, idx, height,
			  width, depth, wsize, grad1);
//backward for right            
  N = input.numel ();
  n = num * channel * height;
  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_right_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g2, height, width, depth, wsize, top_temp);

  hipMemset (top_grad, 0, sizeof (float) * N);
  get_temp_grad <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, grad_out, top_mask, top_grad, 2);

  N = num * channel * width * height;
  MaxDepth <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, height * width, depth, idx);

  sga_right_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g2, top_grad, idx, height, width, depth, wsize,
			  grad_input);
  n = num * channel * width * height;
  sga_right_weight_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, bottom_data, top_temp, top_grad, idx, height,
			  width, depth, wsize, grad2);
}

__global__ void lga_filtering_forward (const int n, const float *bottom_data,
		       const float *filters, const int height,
		       const int width, const int channel, const int radius,
		       float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
//    printf("OK\n");
//    printf("%d, %.2f, %.2f\n", index, bottom_data[index], top_data[index]);
  if (index >= n)
    {
      return;
    }
//    top_data[index]=1.0;
//    assert(0);
  int step = height * width;
  int wsize = 2 * radius + 1;
//      int fsize=wsize*wsize*3;
  int fbase =
    index / (step * channel) * (step * wsize * wsize * 3) + index % step;
  int row = index % step / width;
  int col = index % width;
  int depth = index / step % channel;
  for (int d = -1; d <= 1; d++)
    {
      for (int r = -radius; r <= radius; r++)
	{
	  for (int c = -radius; c <= radius; c++)
	    {
	      int rr = r + row;
	      int cc = c + col;
	      int dd = d + depth;
	      int shift = 0;
	      if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width
		  && dd < channel)
		shift = r * width + c + d * step;
	      int location =
		(d + 1) * (wsize * wsize) + (r + radius) * wsize + c + radius;
	      top_data[index] +=
		bottom_data[index + shift] * filters[fbase + location * step];
	    }
	}
    }
//        top_data[index]=1.0;
//        printf("%d, %d, %d, %.2f, %.2f\n", index, row, col, bottom_data[index], top_data[index]);
}

__global__ void lga_filter_backward (const int n, const float *bottom_data,
		     const float *top_diff, const int height, const int width,
		     const int channel, const int radius, float *filter_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int wsize = 2 * radius + 1;

  int base =
    index / (step * wsize * wsize * 3) * (step * channel) + index % step;
  int location = index / step % (wsize * wsize * 3);
  int d = location / (wsize * wsize) - 1;
  int r = (location / wsize) % wsize - radius;
  int c = location % wsize - radius;

  int rr = index % step / width + r;
  int cc = index % width + c;

  for (int i = 0; i < channel; i++)
    {
      int dd = i + d;
      if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width
	  && dd < channel)
	{
	  int shift = r * width + c + d * step;
	  filter_diff[index] +=
	    top_diff[base + i * step] * bottom_data[base + shift + i * step];
	}
      else
	filter_diff[index] +=
	  top_diff[base + i * step] * bottom_data[base + i * step];
    }


}

__global__ void lga_data_backward (const int n, const float *filters, const float *top_diff,
		   const int height, const int width, const int channel,
		   const int radius, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int wsize = 2 * radius + 1;
//      int fsize=wsize*wsize*3;
  int fbase =
    index / (step * channel) * (step * wsize * wsize * 3) + index % step;
  int row = index % step / width;
  int col = index % width;
  int depth = index / step % channel;
  for (int d = -1; d <= 1; d++)
    {
      for (int r = -radius; r <= radius; r++)
	{
	  for (int c = -radius; c <= radius; c++)
	    {
	      int rr = r + row;
	      int cc = c + col;
	      int dd = d + depth;
	      //      int shift = 0;
	      if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width
		  && dd < channel)
		{
		  int shift = r * width + c + d * step;
		  //      int fshift= r*width+c;
		  int location =
		    (-d + 1) * (wsize * wsize) + (-r + radius) * wsize - c +
		    radius;
		  bottom_diff[index] +=
		    top_diff[index + shift] * filters[fbase + r * width + c +
						      location * step];
		}
	      else
		{
		  int location =
		    (d + 1) * (wsize * wsize) + (r + radius) * wsize + c +
		    radius;
		  bottom_diff[index] +=
		    top_diff[index] * filters[fbase + location * step];
		}
	    }
	}
    }
}

void lga_forward (at::Tensor input, at::Tensor filters, at::Tensor output,
	     const int radius){

//        print_kernel<<<10, 10>>>();
//        hipDeviceSynchronize();
  //       int num=input->size(0);
  int channel = input.size(1);
  int height = input.size(2);
  int width = input.size(3);
  int n = input.numel ();
  //       printf("%d, %d, %d, %d, %d\n", height, width, channel, n, radius);
  //       hipStream_t stream = at::cuda::getCurrentCUDAStream();
/*        float *temp = new float[n];
        float *out = input.data<float>();
        hipMemcpy(temp,out,n*sizeof(float),hipMemcpyDeviceToHost);	
        for(int i=0;i<n;i++)
           printf("%.2f ", temp[i]);
*/
  lga_filtering_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, input.data<float>(), filters.data<float>(),
			  height, width, channel, radius,
			  output.data<float>());
  //     temp = new float[n];


}


void lga_backward (at::Tensor input, at::Tensor filters, at::Tensor gradOutput,
	      at::Tensor gradInput, at::Tensor gradFilters, const int radius){

//      int num=input->size(0);
  int channel = input.size(1);
  int height = input.size(2);
  int width = input.size(3);
//    hipStream_t stream = at::cuda::getCurrentCUDAStream();
  int n = filters.numel ();
  lga_filter_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, input.data<float>(),
			  gradOutput.data<float>(), height, width, channel,
			  radius, gradFilters.data<float>());
//    printf("%d, %d, %d, %d\n", height, width, channel, n);

  n = input.numel ();
  float *grad = gradInput.data<float>();
  hipMemset (grad, 0, sizeof (float) * n);
  lga_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, filters.data<float>(),
			  gradOutput.data<float>(), height, width, channel,
			  radius, grad);

}

void lga3d_forward (at::Tensor input, at::Tensor filters, at::Tensor output,
	       const int radius){

  //       int num=input->size(0);
  int channel = input.size(2);
  int height = input.size(3);
  int width = input.size(4);
  int n = input.numel ();
//        hipStream_t stream = at::cuda::getCurrentCUDAStream();
  lga_filtering_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, input.data<float>(), filters.data<float>(),
			  height, width, channel, radius,
			  output.data<float>());

}


void lga3d_backward (at::Tensor input, at::Tensor filters, at::Tensor gradOutput,
		at::Tensor gradInput, at::Tensor gradFilters,
		const int radius){

//      int num=input->size(0);
  int channel = input.size(2);
  int height = input.size(3);
  int width = input.size(4);
//    hipStream_t stream = at::cuda::getCurrentCUDAStream();
  int n = filters.numel ();
  lga_filter_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, input.data<float>(),
			  gradOutput.data<float>(), height, width, channel,
			  radius, gradFilters.data<float>());

  n = input.numel ();
  float *grad = gradInput.data<float>();
  hipMemset (grad, 0, sizeof (float) * n);
  lga_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, filters.data<float>(),
			  gradOutput.data<float>(), height, width, channel,
			  radius, grad);

}



#ifdef __cplusplus
    }
#endif
